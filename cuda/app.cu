
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* a, int* b, int* c){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
}

__managed__ int vector_a[256], vector_b[256], vector_c[256];

int main(){
    for (int i = 0; i < 256; i++){
        vector_a[i] = i;
        vector_b[i] = 256 - i;
    }

    add<<<1, 256>>>(vector_a, vector_b, vector_c); //<<<blocks, threads per block>>>

    hipDeviceSynchronize();

    int result_sum = 0;

    for (int i = 0; i < 256; i++){
        result_sum += vector_c[i];
    }

    printf("Hi Mom! I added vectors add on GPU");
    printf("Result: sum = %d", result_sum);
}