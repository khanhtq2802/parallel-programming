
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK_ERROR()                                                          \
    do                                                                              \
    {                                                                               \
        hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                     \
        {                                                                           \
            printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
            exit(EXIT_FAILURE);                                                     \
        }                                                                           \
    } while (0)

__global__ void helloCUDA()
{
    printf("Hello CUDA from GPU!\n");
}

int main()
{
    helloCUDA<<<1, 1>>>();
    CUDA_CHECK_ERROR();
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR();
    return 0;
}
