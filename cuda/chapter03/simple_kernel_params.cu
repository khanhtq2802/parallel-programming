#include "hip/hip_runtime.h"
#include "../common/book.h"

__global__ void add(int a, int b, int *c)
{
    *c = a + b; // thay doi gia chi o dia tri duoc pass vao
}

int main(void)
{
    int c;
    int *dev_c;                                             // khoi tao con tro dev_c o host
    HANDLE_ERROR(hipMalloc((void **)&dev_c, sizeof(int))); // pass dia tri cua con tro dev_c vao device

    add<<<1, 1>>>(2, 7, dev_c); // pass gia tri cua dev_c (la dia chi nao do) vao add

    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int),
                            hipMemcpyDeviceToHost)); // pass dia chi cua c, thay doi gia tri cua c = gia tri cua dev_c tro toi
    printf("2 + 7 = %d\n", c);
    HANDLE_ERROR(hipFree(dev_c)); // xoa dev_c khoi device

    return 0;
}